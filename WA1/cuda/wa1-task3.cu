#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>
int timeval_subtract( struct timeval* result, struct timeval* t2, struct timeval* t1) {
        unsigned int resolution=1000000;
        long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec) ;
        result->tv_sec = diff / resolution;
        result->tv_usec = diff % resolution;
        return (diff<0);
}
#define GPU_RUNS 100

__global__ void squareKernel(float* d_in, float *d_out, int sizeN) {
        const unsigned int lid = threadIdx.x; // local id inside a block
        const unsigned int gid = blockIdx.x*blockDim.x + lid; // global id
        if(gid<sizeN){
                d_out[gid] = powf((d_in[gid]/(d_in[gid]-2.3)), 3); // do computation
        }
}

int main(int argc, char** argv) {
        unsigned int N = 753411;
        unsigned int mem_size = N*sizeof(float);
        unsigned int block_size = 256;
        unsigned int num_blocks = ((N + (block_size - 1)) / block_size);

        // allocate host memory
        float* h_in = (float*) malloc(mem_size);
        float* h_out = (float*) malloc(mem_size);
        // initialize the memory
        for(unsigned int i=0; i<N; ++i){
                h_in[i] = (float)i;
        }
        // allocate device memory
        float* d_in;
        float* d_out;
        hipMalloc((void**)&d_in, mem_size);
        hipMalloc((void**)&d_out, mem_size);
        // copy host memory to device
        hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        // execute the kernel
        for(int i=0; i<GPU_RUNS; i++) {
                squareKernel<<< num_blocks, block_size>>>(d_in, d_out, N);
        }
        hipDeviceSynchronize();


        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / GPU_RUNS;
        printf("Took %d microseconds (%.2fms)nn",elapsed,elapsed/1000.0);


        // copy result from ddevice to host
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
        // print result
        //for(unsigned int i=0; i<N; ++i) printf("%.6fnn", h_out[i]);
        // clean-up memory
        free(h_in); free(h_out);
        hipFree(d_in); hipFree(d_out);
}